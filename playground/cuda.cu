#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#define N (2048*2048)
#define THREADS_PER_BLOCK 512
using namespace std;

__global__ void mykernal() { // Run on device and called by host

}

__global__ void add(int *a, int *b, int *c, int n) {
    // *c = *a + *b;
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) c[index] = a[index] + b[index];
}

void random_ints(int* x, int size) {
	for (int i = 0; i < size; i++) {
		x[i] = rand() % 10;
	}
}

int main() {
    int nDevices;

    hipGetDeviceCount(&nDevices);
    cout << nDevices << endl;
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
            prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
            prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
            2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = (int *)malloc(size); random_ints(a, N);
    b = (int *)malloc(size); random_ints(b, N);
    c = (int *)malloc(size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    
    add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0;i < 10; i++) {
		printf("a[%d]=%d , b[%d]=%d, c[%d]=%d\n",i,a[i],i,b[i],i,c[i]);
	}
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}
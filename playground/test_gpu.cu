#include <iostream>
#include <vector>
#include <string>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#define N (2048*2048)
#define THREADS_PER_BLOCK 1024
using namespace std;

class Cell {
public:
    int val;
    char* cs;
    Cell() { val = rand() % 10; }
    Cell(int v): val(v) {}
    ~Cell() {};
    __host__ __device__ void setVal(int v) {
        val = v;
        // cs = "B";
        // vec[0] = v;
    }
    __host__ __device__ int getVal() const {
        return val;
    }
    void print() { cout << val << " "; }
};

__global__ void add(Cell *a, Cell *b, Cell *c, int n) {
    // *c = *a + *b;
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) c[index].setVal(a[index].getVal() + b[index].getVal());
}


int main() {
    cout << "Size of Cell: " << sizeof(Cell) << endl;

    Cell *a, *b, *c;
    Cell *d_a, *d_b, *d_c;
    int size = N * sizeof(Cell);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // a = (Cell*)malloc(size);
    // b = (Cell*)malloc(size);
    // c = (Cell*)malloc(size);
    a = new Cell [N];
    b = new Cell [N];
    c = new Cell [N];

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; ++i) {
        a[i].print();
        cout << " ";
        b[i].print();
        cout << " ";
        c[i].print();
        cout << endl;
    }

    delete [] a;
    delete [] b;
    delete [] c;
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}